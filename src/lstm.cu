#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hipblas.h>
#include <hiprand.h>

#define TRAINING (false)

#define GROUP_GEMM 1
#define USE_GEMM_STREAMS 0
#define FUSE_PW 1
#define PRE_TRANSPOSE 1
#define RECUR_BATCH_SIZE 2
#define USE_LAYERS_STREAMS 1

// Define some error checking macros.
#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
    if (stat != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
    }
}

#define cublasErrCheck(stat) { cublasErrCheck_((stat), __FILE__, __LINE__); }
void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line) {
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
    }
}

#define curandErrCheck(stat) { curandErrCheck_((stat), __FILE__, __LINE__); }
void curandErrCheck_(hiprandStatus_t stat, const char *file, int line) {
    if (stat != HIPRAND_STATUS_SUCCESS) {
        fprintf(stderr, "cuRand Error: %d %s %d\n", stat, file, line);
    }
}


// Device functions
__forceinline__ __device__ float sigmoidf(float in) {
    return 1.f / (1.f + expf(-in));  
}

// Pointwise functions
__global__ void pw_biasAdd(float *y, float *bias, int n, int nBias) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) y[i] += bias[i % nBias];
}

__global__ void pw_vecAdd(float *y, float *a,  float *b, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) y[i] = a[i] + b[i];
}

__global__ void pw_vecMul(float *y, float *a,  float *b, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) y[i] = a[i] * b[i];
}

__global__ void pw_tanh(float *y, float *a, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) y[i] = tanh(a[i]);
}

__global__ void pw_sigmoid(float *y, float *a, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) y[i] = sigmoidf(a[i]);
}

// Unfused LSTM (calling many pointwise kernels).
int LSTM_unit_unfused(int hiddenSize, 
                            int miniBatch,
                            float * __restrict__ h_in, // h(t-1) * R
                            float * __restrict__ x_in, // x(t) * W
                            float * __restrict__ bias,
                            float * __restrict__ h_out,// h(t)
                            float * __restrict__ c_in, // c(t-1)
                            float * __restrict__ c_out,// c(t)
                            hipStream_t stream) {
    dim3 blockDim, gridDim;
    
    int numElements = hiddenSize * miniBatch;
    
    blockDim.x = 128;
    gridDim.x = (numElements + blockDim.x - 1) / blockDim.x;

    // element wise calculations
    // x(t) = x(t) * W + h(t-1) * R + bias, as input to this unit
    for (int i = 0; i < 4; i++) {
        pw_vecAdd <<< gridDim, blockDim, 0, stream >>> (x_in + i * numElements, 
                                                        x_in + i * numElements, 
                                                        h_in + i * numElements,
                                                        numElements);
        cudaErrCheck(hipGetLastError());

        pw_biasAdd <<< gridDim, blockDim, 0, stream >>> (x_in + i * numElements, 
                                                         bias + i * hiddenSize, 
                                                         numElements, 
                                                         hiddenSize);
        cudaErrCheck(hipGetLastError());
        
        pw_biasAdd <<< gridDim, blockDim, 0, stream >>> (x_in + i * numElements, 
                                                         bias + (i + 4) * hiddenSize, 
                                                         numElements, 
                                                         hiddenSize);
        cudaErrCheck(hipGetLastError());
    }    
    
    // x(t) goes through 4 gates' activation
    pw_sigmoid <<< gridDim, blockDim, 0, stream >>> (x_in + 0 * numElements, x_in + 0 * numElements, numElements);
    cudaErrCheck(hipGetLastError());
    
    pw_sigmoid <<< gridDim, blockDim, 0, stream >>> (x_in + 1 * numElements, x_in + 1 * numElements, numElements);
    cudaErrCheck(hipGetLastError());
    
    pw_tanh <<< gridDim, blockDim, 0, stream >>> (x_in + 2 * numElements, x_in + 2 * numElements, numElements);
    cudaErrCheck(hipGetLastError());
    
    pw_sigmoid <<< gridDim, blockDim, 0, stream >>> (x_in + 3 * numElements, x_in + 3 * numElements, numElements);
    cudaErrCheck(hipGetLastError());
    
    // assign location to 4 gates
    float *in_gate      = x_in + 0 * numElements;
    float *forget_gate = x_in + 1 * numElements;
    float *in_gate2     = x_in + 2 * numElements;
    float *out_gate     = x_in + 3 * numElements;
    
    // f(t) *= c(t-1)
    pw_vecMul <<< gridDim, blockDim, 0, stream >>> (forget_gate, forget_gate, c_in, numElements);
    cudaErrCheck(hipGetLastError());

    // i(t) *= g(t)
    pw_vecMul <<< gridDim, blockDim, 0, stream >>> (in_gate, in_gate, in_gate2, numElements);
    cudaErrCheck(hipGetLastError());

    // i(t) += f(t)  
    pw_vecAdd <<< gridDim, blockDim, 0, stream >>> (in_gate, in_gate, forget_gate, numElements);
    cudaErrCheck(hipGetLastError());

    // c(t) = i(t), output cell state
    cudaErrCheck(hipMemcpyAsync(c_out, in_gate, numElements * sizeof(float), hipMemcpyDeviceToDevice, stream));
    
    // i(t) = tanh(i(t)), i(t) === c(t) here, but we must not modify c(t)
    pw_tanh <<< gridDim, blockDim, 0, stream >>> (in_gate, in_gate, numElements);
    cudaErrCheck(hipGetLastError());
    
     // h(t) = i(t) * o(t)
    pw_vecMul <<< gridDim, blockDim, 0, stream >>> (h_out, out_gate, in_gate, numElements);
    cudaErrCheck(hipGetLastError());
    
    return 0;
}

__global__ void LSTM_unit_fused(int hiddenSize,
                                int miniBatch,
                                float * __restrict__ h_in,
                                float * __restrict__ x_in,
                                float * __restrict__ bias,
                                float * __restrict__ linearGates,
                                float * __restrict__ h_out,
                                float * __restrict__ c_in,
                                float * __restrict__ c_out,
                                bool training) {
    
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int numElements = miniBatch * hiddenSize;

    if (index >= numElements) return;

    int currentBatch = index / hiddenSize;
    int gateIndex = (index % hiddenSize) + 4 * currentBatch * hiddenSize;

    float gate[4];

    for (int i = 0; i < 4; i++) {
        gate[i] = x_in[i * hiddenSize + gateIndex] + h_in[i * hiddenSize + gateIndex];
        gate[i] += bias[i * hiddenSize + index % hiddenSize] + bias[(i + 4) * hiddenSize + index % hiddenSize];

        if (training) linearGates[gateIndex + i * hiddenSize] = gate[i];
    }

    float in_gate = sigmoidf(gate[0]);
    float forget_gate = sigmoidf(gate[1]);
    float in_gate2 = tanhf(gate[2]);
    float out_gate = sigmoidf(gate[3]);

    float value = (c_in[index] * forget_gate) + (in_gate * in_gate2);

    c_out[index] = value;

    value = out_gate * tanhf(value);

    h_out[index] = value;
}


float LSTMTest(int hiddenSize, int miniBatch, int seqLength, int numLayers, bool checkF) {
    int numElements = hiddenSize * miniBatch;

    // alloc device memory
    float *h_data, *x_data, *c_data;
    cudaErrCheck(hipMalloc((void**)&h_data, (seqLength + 1) * (numLayers) * numElements * sizeof(float)));
    cudaErrCheck(hipMalloc((void**)&x_data, (seqLength) * (numLayers + 1) * numElements * sizeof(float)));
    cudaErrCheck(hipMalloc((void**)&c_data, (seqLength + 1) * (numLayers) * numElements * sizeof(float)));
    
    float *weight, *weight_T;
    cudaErrCheck(hipMalloc((void**)&weight, numLayers * hiddenSize * hiddenSize * 8 * sizeof(float)));
    cudaErrCheck(hipMalloc((void**)&weight_T, numLayers * hiddenSize * hiddenSize * 8 * sizeof(float)));
    
    float *bias;
    cudaErrCheck(hipMalloc((void**)&bias, numLayers * hiddenSize * 8 * sizeof(float)));
    
    float *h_in, *x_in;
    cudaErrCheck(hipMalloc((void**)&h_in, 4 * numLayers * numElements * sizeof(float)));
    cudaErrCheck(hipMalloc((void**)&x_in, 4 * seqLength * numElements * sizeof(float)));

    float *linearGates;
    // Activations
    if (TRAINING) {
        cudaErrCheck(hipMalloc((void**)&linearGates, 4 * seqLength * numLayers * numElements * sizeof(float)));
    }
    
    // operation wise streams for optimization 2
    hipStream_t stream_x_single, stream_h_single;

    // (operation + layer) wise streams for optimization 6
    hipStream_t *stream_x, *stream_h;
    stream_x = (hipStream_t*)malloc(numLayers * sizeof(hipStream_t));
    stream_h = (hipStream_t*)malloc(numLayers * sizeof(hipStream_t));

    if (USE_GEMM_STREAMS) {
        // optimization 2 uses different streams for x and h
        cudaErrCheck(hipStreamCreate(&stream_x_single));
        cudaErrCheck(hipStreamCreateWithPriority(&stream_h_single, 0, -1));   
    }
    else {
        for (int i = 0; i < numLayers; i++) {
            if (USE_LAYERS_STREAMS) {
                // optimization 6 uses different streams for various layers
                cudaErrCheck(hipStreamCreate(&stream_x[i]));
                cudaErrCheck(hipStreamCreateWithPriority(&stream_h[i], 0, -1));   
            }
            else {
                stream_x[i] = NULL;  
                stream_h[i] = NULL;  
            }
        }
    }
    
    // alloc events
    hipEvent_t **events_x, **events_h;
    events_x = (hipEvent_t**)malloc(numLayers * sizeof(hipEvent_t*));
    events_h = (hipEvent_t**)malloc(numLayers * sizeof(hipEvent_t*));
    for (int i = 0; i < numLayers; i++) {
        events_x[i] = (hipEvent_t*)malloc(seqLength * sizeof(hipEvent_t));
        events_h[i] = (hipEvent_t*)malloc(seqLength * sizeof(hipEvent_t));
    }

    // initiate random inputs
    hiprandGenerator_t gen;
    curandErrCheck(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
    curandErrCheck(hiprandSetPseudoRandomGeneratorSeed(gen, 1782ULL));
    curandErrCheck(hiprandGenerateUniform(gen, h_data, (seqLength + 1) * (numLayers) * numElements));
    curandErrCheck(hiprandGenerateUniform(gen, c_data, (seqLength + 1) * (numLayers) * numElements));
    curandErrCheck(hiprandGenerateUniform(gen, x_data, (seqLength) * (numLayers + 1) * numElements));
    curandErrCheck(hiprandGenerateUniform(gen, weight, numLayers * hiddenSize * hiddenSize * 8));
    curandErrCheck(hiprandGenerateUniform(gen, bias, numLayers * hiddenSize * 8));
    curandErrCheck(hiprandDestroyGenerator(gen));

    // create cuBLAS handle.
    hipblasHandle_t handle;
    cublasErrCheck(hipblasCreate(&handle));
    
    cudaErrCheck(hipDeviceSynchronize());
  
    // start timing
    float elapsedTime;
    hipEvent_t start, stop;
    cudaErrCheck(hipEventCreate(&start));
    cudaErrCheck(hipEventCreate(&stop));
    cudaErrCheck(hipEventRecord(start));
  
    // LSTM

    const hipblasOperation_t a_trans = (PRE_TRANSPOSE && (seqLength > 1)) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    const hipblasOperation_t b_trans = HIPBLAS_OP_N; // always N
    
    // hipblasSgemm(): C = alpha * (A + B) + beta * C 
    float alpha = 1.f;
    float beta  = 0.f; 
           
    if (a_trans == HIPBLAS_OP_N) {        
        // do optimization 4 here, transpose A
        for (int layer = 0; layer <numLayers; layer++) {

            // determine whether using same streams among layers
            hipStream_t stream_x_this_layer, stream_h_this_layer;
            if (USE_GEMM_STREAMS) {
                stream_x_this_layer = stream_x_single;
                stream_h_this_layer = stream_h_single;
            }
            else {
                stream_x_this_layer = stream_x[layer];
                stream_h_this_layer = stream_h[layer];
            }

            // for x(t)
            float *W_weight_in = weight + layer * hiddenSize * hiddenSize * 8;
            float *W_weight_out = weight_T + layer * hiddenSize * hiddenSize * 8;

            // for h(t-1)
            float *R_weight_in = weight + layer * hiddenSize * hiddenSize * 8 + hiddenSize * hiddenSize * 4;
            float *R_weight_out = weight_T + layer *hiddenSize * hiddenSize * 8 + hiddenSize * hiddenSize * 4;

            cublasErrCheck(hipblasSetStream(handle, stream_x_this_layer));
            cublasErrCheck(hipblasSgeam(handle, HIPBLAS_OP_T, // trans A
                                        HIPBLAS_OP_N, // trans B
                                        4 * hiddenSize, // #rows in A & C
                                        hiddenSize, // #cols in B & C
                                        &alpha, // scale A
                                        W_weight_in, // A
                                        hiddenSize, // leading dim in A
                                        &beta, // scale B
                                        NULL, // B
                                        4 * hiddenSize, // leading dim in B
                                        W_weight_out, // C
                                        4 * hiddenSize)); // leading dim in C
            
            cublasErrCheck(hipblasSetStream(handle, stream_h_this_layer));
            cublasErrCheck(hipblasSgeam(handle, HIPBLAS_OP_T, // trans A
                                        HIPBLAS_OP_N, // trans B
                                        4 * hiddenSize, // #rows in A & C
                                        hiddenSize, // #cols in B & C
                                        &alpha, // scale A
                                        R_weight_in, // A
                                        hiddenSize, // leading dim in A
                                        &beta, // scale B
                                        NULL, // B
                                        4 * hiddenSize, // leading dim in B
                                        R_weight_out, // C
                                        4 * hiddenSize)); // leading dim in C
        }
    }
    else {
        weight_T = weight;
    }

    
    int lStart = 0; // layer starts from
    int lEnd = 0;   // layer ends at
    int tStart = 0; // timestep starts from 
    int tEnd = 0;   // timestep ends at
    int recurBatchSize = RECUR_BATCH_SIZE; // optimization 5 will make it 2
    
    while (true) {
        // Many layer "scheduling".
        if (lEnd == 0) {
            lStart = 0;
            lEnd = 1;
            tStart = 0;
        }
        else {
            // Move "up" and "left"
            lStart++;
            lEnd++;
            
            tStart -= recurBatchSize;
            
            // Over the top or off the left, reset to layer 0
            if (lEnd > numLayers || tStart < 0) {
                tStart += (lStart + 1) * recurBatchSize;

                lStart = 0;
                lEnd = 1;
            }
            
            // Off the right, step up
            while (tStart >= seqLength && lEnd <= numLayers) {
                lStart++;
                lEnd++;
                
                tStart -= recurBatchSize;
            }
            
            
            // Over the top or off the left, done!
            if (lEnd > numLayers || tStart < 0) {
                break;
            }
        }
        
        tEnd = tStart + recurBatchSize;
        if (tEnd > seqLength) tEnd = seqLength;
        
        // lStart, lEnd always differ 1
        for (int layer = lStart; layer < lEnd; layer++) {

            // determine whether using same streams among layers
            hipStream_t stream_x_this_layer, stream_h_this_layer;
            if (USE_GEMM_STREAMS) {
                stream_x_this_layer = stream_x_single;
                stream_h_this_layer = stream_h_single;
            }
            else {
                stream_x_this_layer = stream_x[layer];
                stream_h_this_layer = stream_h[layer];
            }

            // do x(t) * W_weight on stream_x[layer]
            cublasErrCheck(hipblasSetStream(handle, stream_x_this_layer));
            
            // tStart, tEnd differ recurBatchSize
            for (int i = tStart; i < tEnd; i++) {
                if (layer > 0) {
                    cudaErrCheck(hipStreamWaitEvent(stream_x_this_layer, events_h[layer - 1][i], 0));
                    cudaErrCheck(hipEventDestroy(events_h[layer - 1][i]));
                }
            }

            // x(t) *= [W_weight]
            if (GROUP_GEMM) {
                // do optimization 1 here
                cublasErrCheck(hipblasSgemm(handle,
                                    a_trans, b_trans,
                                    4 * hiddenSize, // #rows of A and C
                                    miniBatch * (tEnd - tStart), // #cols of B and C
                                    hiddenSize, // #cols of A and B
                                    &alpha,
                                    &weight_T[layer * 8 * hiddenSize * hiddenSize], // A
                                    a_trans == HIPBLAS_OP_N ? 4 * hiddenSize : hiddenSize, // leading dimension of A, where we can try different data layout
                                    x_data + tStart * numElements + layer * seqLength * numElements, // B
                                    hiddenSize, // leading dimension of B, where we can try different data layout
                                    &beta,
                                    x_in + 4 * tStart * numElements, // C
                                    4 * hiddenSize // leading dimension of C
                                    )); 
            }
            else {
                for (int igemm =0; igemm < 4; igemm++) {
                    cublasErrCheck(hipblasSgemm(handle,
                                    a_trans, b_trans,
                                    hiddenSize, // #rows of A and C
                                    miniBatch * (tEnd - tStart), // #cols of B and C
                                    hiddenSize, // #cols of A and B
                                    &alpha,
                                    &weight_T[layer * 8 * hiddenSize * hiddenSize + igemm * hiddenSize], // A
                                    a_trans == HIPBLAS_OP_N ? 4 * hiddenSize : hiddenSize, // leading dimension of A, where we can try different data layout
                                    x_data + tStart * numElements + layer * seqLength * numElements, // B
                                    hiddenSize, // leading dimension of B, where we can try different data layout
                                    &beta,
                                    x_in + 4 * tStart * numElements + igemm * hiddenSize, // C
                                    4 * hiddenSize // leading dimension of C
                                    )); 
                }
            }
            
            for (int i = tStart; i < tEnd; i++) {
                cudaErrCheck(hipEventCreate(&events_x[layer][i], hipEventDisableTiming));
                cudaErrCheck(hipEventRecord(events_x[layer][i], stream_x_this_layer));  
            }                
            
            for (int i = tStart; i < tEnd; i++) {
                // do h(t-1) *= [R_weight] on stream_h[layer]
                cublasErrCheck(hipblasSetStream(handle, stream_h_this_layer));

                // h(t-1) *= [R_weight]
                if (GROUP_GEMM) {
                     // do optimization 1 here
                     cublasErrCheck(hipblasSgemm(handle,
                                        a_trans, b_trans,
                                        4 * hiddenSize, miniBatch, hiddenSize,
                                        &alpha,
                                        &weight_T[4 * hiddenSize * hiddenSize + layer * 8 * hiddenSize * hiddenSize], 
                                        a_trans == HIPBLAS_OP_N ? 4 * hiddenSize : hiddenSize,
                                        h_data + i * numElements + layer * (seqLength + 1) * numElements,
                                        hiddenSize,
                                        &beta,
                                        h_in + 4 * layer * numElements, 
                                        4 * hiddenSize));
                }
                else {
                    for (int igemm =0; igemm < 4; igemm++) {
                        cublasErrCheck(hipblasSgemm(handle,
                                        a_trans, b_trans,
                                        hiddenSize, miniBatch, hiddenSize,
                                        &alpha,
                                        &weight_T[4 * hiddenSize * hiddenSize + layer * 8 * hiddenSize * hiddenSize + igemm * hiddenSize], 
                                        a_trans == HIPBLAS_OP_N ? 4 * hiddenSize : hiddenSize,
                                        h_data + i * numElements + layer * (seqLength + 1) * numElements,
                                        hiddenSize,
                                        &beta,
                                        h_in + 4 * layer * numElements + igemm * hiddenSize, 
                                        4 * hiddenSize));
                    }
                }

                cudaErrCheck(hipStreamWaitEvent(stream_h_this_layer, events_x[layer][i], 0));
                cudaErrCheck(hipEventDestroy(events_x[layer][i]));

                if (FUSE_PW) {
                    // optimization 3
                    dim3 blockDim, gridDim;

                    blockDim.x = 256;
                    gridDim.x = (numElements + blockDim.x - 1) / blockDim.x;

                    LSTM_unit_fused <<< gridDim, blockDim, 0, stream_h_this_layer >>>
                            (hiddenSize, miniBatch,
                            h_in + 4 * layer * numElements,
                            x_in + 4 * i * numElements,
                            bias + 8 * layer * hiddenSize,
                            TRAINING ? linearGates + 4 * (i * numElements + layer * seqLength * numElements) : NULL,
                            h_data + (i + 1) * numElements + layer * (seqLength + 1) * numElements,
                            c_data + i * numElements + layer * (seqLength + 1) * numElements,
                            c_data + (i + 1) * numElements + layer * (seqLength + 1) * numElements,
                            TRAINING);
                    cudaErrCheck(hipGetLastError());
                }
                else {
                    LSTM_unit_unfused(hiddenSize, miniBatch,
                            h_in + 4 * layer * numElements, 
                            x_in + 4 * i * numElements,
                            bias + 8 * layer * hiddenSize,
                            h_data + (i + 1) * numElements + layer * (seqLength + 1) * numElements,
                            c_data + i * numElements + layer * (seqLength + 1) * numElements,
                            c_data + (i + 1) * numElements + layer * (seqLength + 1) * numElements,
                            stream_h_this_layer);
                }
                if (layer != numLayers - 1) {
                    cudaErrCheck(hipEventCreate(&events_h[layer][i], hipEventDisableTiming));
                    cudaErrCheck(hipEventRecord(events_h[layer][i], stream_h_this_layer));  
                }
            }
        }
    } 

    // stop timing
    cudaErrCheck(hipEventRecord(stop));
    cudaErrCheck(hipEventSynchronize(stop));
    cudaErrCheck(hipEventElapsedTime(&elapsedTime, start, stop));
    cudaErrCheck(hipDeviceSynchronize());

    // free everything
    cudaErrCheck(hipFree(h_data));
    cudaErrCheck(hipFree(x_data));  
    cudaErrCheck(hipFree(c_data));  

    if (weight != weight_T) cudaErrCheck(hipFree(weight)); 
    cudaErrCheck(hipFree(weight_T));
    
    cudaErrCheck(hipFree(bias));
    
    cudaErrCheck(hipFree(h_in));
    cudaErrCheck(hipFree(x_in));
    if (TRAINING) cudaErrCheck(hipFree(linearGates));

    if (USE_GEMM_STREAMS) {
        cudaErrCheck(hipStreamDestroy(stream_x_single));
        cudaErrCheck(hipStreamDestroy(stream_h_single));
    }
    else {
        for (int i = 0; i < numLayers; i++) {
            if (stream_x[i] != NULL) cudaErrCheck(hipStreamDestroy(stream_x[i]));
            if (stream_h[i] != NULL) cudaErrCheck(hipStreamDestroy(stream_h[i]));
        }
    }

    free(stream_x);
    free(stream_h);
    
    for (int i = 0; i < numLayers; i++) {
        free(events_x[i]);
        free(events_h[i]);
    }
    free(events_x);
    free(events_h);
    
    return elapsedTime;
}


int main(int argc, char* argv[]) {
    int seqLength;
    int numLayers;
    int hiddenSize;
    int miniBatch; 
    
    if (argc == 5) {
        seqLength = atoi(argv[1]);
        numLayers =  atoi(argv[2]);
        hiddenSize =  atoi(argv[3]);
        miniBatch =  atoi(argv[4]);    
    }
    else if (argc == 1) {
        printf("Running with default settings\n");
        seqLength = 100;
        numLayers = 4;
        hiddenSize = 512;
        miniBatch = 64;
    }
    else {
        printf("Usage: ./LSTM <seqLength> <numLayers> <hiddenSize> <miniBatch>\n");
        return 1;        
    }

    printf("seqLength %d, numLayers %d, hiddenSize %d, miniBatch %d\n", seqLength, numLayers, hiddenSize, miniBatch);  
    
    int numRuns = 1;
    
    float totalTime = 0.f;
    for (int run = 0; run < numRuns; run++) {
        totalTime += LSTMTest(hiddenSize, miniBatch, seqLength, numLayers, true);
    }
    
    printf("Runtime %fms\n", totalTime / numRuns);
    
    return time < 0;
}